#include "hip/hip_runtime.h"
#include "kernel.h"
#include "tensor.h"

#include <stdexcept>

__global__ void gemm(__hip_bfloat16 *__restrict__ out,
                     const __hip_bfloat16 *__restrict__ in_a,
                     const __hip_bfloat16 *__restrict__ in_b,
                     const __hip_bfloat16 *__restrict__ bias,
                     __hip_bfloat16 scale, std::size_t m, std::size_t n,
                     std::size_t k) {
  const auto row = blockIdx.x * blockDim.x + threadIdx.x;
  const auto col = blockIdx.y * blockDim.y + threadIdx.y;
  if (row < m && col < n) {
    auto res = bias ? bias[row * n + col] : __hip_bfloat16{0};
    for (int i = 0; i < k; i++)
      res += scale * in_a[row * k + i] * in_b[i * n + col];
    out[row * n + col] = res;
  }
}

__global__ void gemm_transposed(__hip_bfloat16 *__restrict__ out,
                                const __hip_bfloat16 *__restrict__ in_a,
                                const __hip_bfloat16 *__restrict__ in_b,
                                const __hip_bfloat16 *__restrict__ bias,
                                __hip_bfloat16 scale, std::size_t m,
                                std::size_t n, std::size_t k) {
  const auto row = blockIdx.x * blockDim.x + threadIdx.x;
  const auto col = blockIdx.y * blockDim.y + threadIdx.y;
  if (row < m && col < n) {
    auto res = bias ? bias[row * n + col] : __hip_bfloat16{0};
    for (int i = 0; i < k; i++)
      res += scale * in_a[row * k + i] * in_b[col * k + i];
    out[row * n + col] = res;
  }
}

void launch_gemm(Tensor &out, const Tensor &in_a, const Tensor &in_b,
                 const Tensor &bias, __hip_bfloat16 scale,
                 bool transpose_second) {
  if (in_a.dimensions != 2 || in_b.dimensions != 2 || out.dimensions != 2)
    throw std::runtime_error("invalid dimension");
  const auto m = in_a.shape[0];
  const auto k = in_a.shape[1];
  const auto n = transpose_second ? in_b.shape[0] : in_b.shape[1];
  if (k != in_b.shape[transpose_second ? 1 : 0])
    throw std::runtime_error("incompatible dimension");
  if (m != out.shape[0] || n != out.shape[1])
    throw std::runtime_error("incompatible dimension");

  const dim3 threads_per_block(16, 16);
  const dim3 num_blocks((m + 15) / 16, (n + 15) / 16);
  if (transpose_second) {
    gemm_transposed<<<num_blocks, threads_per_block>>>(
        out.storage->data, in_a.storage->data, in_b.storage->data,
        bias.storage->data, scale, m, n, k);
  } else {
    gemm<<<num_blocks, threads_per_block>>>(
        out.storage->data, in_a.storage->data, in_b.storage->data,
        bias.storage->data, scale, m, n, k);
  }
}
