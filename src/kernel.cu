#include "hip/hip_runtime.h"
#include "kernel.h"
#include "tensor.h"

#include <stdexcept>

__global__ void gemm(__hip_bfloat16 *__restrict__ out,
                     const __hip_bfloat16 *__restrict__ in_a,
                     const __hip_bfloat16 *__restrict__ in_b,
                     const __hip_bfloat16 *__restrict__ bias,
                     __hip_bfloat16 scale, std::size_t m, std::size_t n,
                     std::size_t k) {
  const auto row = blockIdx.x * blockDim.x + threadIdx.x;
  const auto col = blockIdx.y * blockDim.y + threadIdx.y;
  if (row < m && col < n) {
    auto res = bias ? bias[col] : __hip_bfloat16{0};
    for (int i = 0; i < k; i++)
      res += scale * in_a[row * k + i] * in_b[i * n + col];
    out[row * n + col] = res;
  }
}

__global__ void gemm_transposed(__hip_bfloat16 *__restrict__ out,
                                const __hip_bfloat16 *__restrict__ in_a,
                                const __hip_bfloat16 *__restrict__ in_b,
                                const __hip_bfloat16 *__restrict__ bias,
                                __hip_bfloat16 scale, std::size_t m,
                                std::size_t n, std::size_t k) {
  const auto row = blockIdx.x * blockDim.x + threadIdx.x;
  const auto col = blockIdx.y * blockDim.y + threadIdx.y;
  if (row < m && col < n) {
    auto res = bias ? bias[col] : __hip_bfloat16{0};
    for (int i = 0; i < k; i++)
      res += scale * in_a[row * k + i] * in_b[col * k + i];
    out[row * n + col] = res;
  }
}

__global__ void dense(__hip_bfloat16 *__restrict__ out,
                      const __hip_bfloat16 *__restrict__ x,
                      const __hip_bfloat16 *__restrict__ weight,
                      const __hip_bfloat16 *__restrict__ bias, std::size_t n,
                      std::size_t in_features, std::size_t out_features) {
  const auto row = blockIdx.x * blockDim.x + threadIdx.x;
  const auto col = blockIdx.y * blockDim.y + threadIdx.y;
  if (row < n && col < out_features) {
    auto res = bias ? bias[col] : __hip_bfloat16{0};
    for (int i = 0; i < in_features; i++)
      res += x[row * in_features + i] * weight[col * in_features + i];
    const auto res_f32 = __bfloat162float(res);
    out[row * out_features + col] =
        __float2bfloat16(res_f32 / (1.0f + __expf(-res_f32)));
  }
}

void launch_gemm(Tensor &out, const Tensor &in_a, const Tensor &in_b,
                 const Tensor &bias, __hip_bfloat16 scale,
                 bool transpose_second) {
  if (in_a.dimensions != 2 || in_b.dimensions != 2 || out.dimensions != 2)
    throw std::runtime_error("invalid dimension");
  const auto m = in_a.shape[0];
  const auto k = in_a.shape[1];
  const auto n = transpose_second ? in_b.shape[0] : in_b.shape[1];
  if (k != in_b.shape[transpose_second ? 1 : 0])
    throw std::runtime_error("incompatible dimension");
  if (m != out.shape[0] || n != out.shape[1])
    throw std::runtime_error("incompatible dimension");

  const dim3 threads_per_block(16, 16);
  const dim3 num_blocks((m + 15) / 16, (n + 15) / 16);
  if (transpose_second) {
    gemm_transposed<<<num_blocks, threads_per_block>>>(
        out.storage->data, in_a.storage->data, in_b.storage->data,
        bias.storage->data, scale, m, n, k);
  } else {
    gemm<<<num_blocks, threads_per_block>>>(
        out.storage->data, in_a.storage->data, in_b.storage->data,
        bias.storage->data, scale, m, n, k);
  }
}
